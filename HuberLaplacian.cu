#include "hip/hip_runtime.h"
/**
 *   ___ _   _ ___   _     __  __   _   ___     
 *  / __| | | |   \ /_\   |  \/  | /_\ | _ \    
 * | (__| |_| | |) / _ \  | |\/| |/ _ \|  _/    
 *  \___|\___/|___/_/_\_\_|_|__|_/_/_\_\_|_ ___ 
 *       / __| | | | _ \ __| _ \___| _ \ __/ __|
 *       \__ \ |_| |  _/ _||   /___|   / _|\__ \
 *       |___/\___/|_| |___|_|_\   |_|_\___|___/
 *                                          2012
 *
 *   by Jens Wetzl           (jens.wetzl@fau.de)
 *  and Oliver Taubmann (oliver.taubmann@fau.de)
 *
 *  This work is licensed under a Creative Commons
 *  Attribution 3.0 Unported License. (CC-BY)
 *  http://creativecommons.org/licenses/by/3.0/
 * 
 **/

#include "HuberLaplacian.h"
#include "Reduction.h"

#include "CudaLBFGS/error_checking.h"

namespace gpu_HuberLaplacian
{
	__global__ void laplacian(float *dst, const float *src, const size_t width, const size_t height, 
	                    const size_t pixelsPerThread);
	__global__ void huber(float *x, const size_t width, const size_t height, const float alpha, 
	                      const float strength, const size_t pixelsPerThread, float *f);

	float *d_tmp;
}

HuberLaplacian::HuberLaplacian(const size_t height, const size_t width,
                               const float alpha, const float strength)
   : cost_function(height * width)
   , m_height(height)
   , m_width(width)
   , m_alpha(alpha)
   , m_strength(strength)
{
	CudaSafeCall( hipMalloc(&gpu_HuberLaplacian::d_tmp, m_numDimensions * sizeof(float)) );
	CudaSafeCall( hipMalloc( (void**) &m_reductionArray,  width * height * sizeof(float)) );
	CudaSafeCall( hipMalloc( (void**) &m_reductionArray2, 1024 * sizeof(float)) );
	
#ifdef SUPERRES_TIMING
	m_atomic = new timer("priorOther");
	m_filter = new timer("priorFilter");
#endif
}

HuberLaplacian::~HuberLaplacian()
{
	CudaSafeCall( hipFree(gpu_HuberLaplacian::d_tmp) );
	CudaSafeCall( hipFree(m_reductionArray)    );
	CudaSafeCall( hipFree(m_reductionArray2)   );
	
#ifdef SUPERRES_TIMING
	m_atomic->saveMeasurement();
	m_filter->saveMeasurement();
	
	delete m_atomic;
	delete m_filter;
#endif
}

void HuberLaplacian::f_gradf(const float *d_x, float *d_f, float *d_gradf)
{
	using namespace gpu_HuberLaplacian;

	dim3 blockDim(512);
	
	const size_t pixelsPerThread = 8;
	size_t threadsPerColumn = (m_height % pixelsPerThread == 0) ? (m_height / pixelsPerThread)
	                                                            : (m_height / pixelsPerThread) + 1;
	size_t threads = threadsPerColumn * m_width;
	
	dim3 gridDim  = (threads % blockDim.x == 0) ? (threads / blockDim.x)
		                                        : (threads / blockDim.x) + 1;
	
#ifdef SUPERRES_TIMING
	m_filter->start();
#endif

	// Compute image Laplacian
	laplacian<<<gridDim, blockDim>>>(d_tmp, d_x, m_width, m_height, pixelsPerThread);

	CudaCheckError();
	hipDeviceSynchronize();
	
#ifdef SUPERRES_TIMING
	m_filter->stop();
	m_atomic->start();
#endif

	CudaSafeCall( hipMemset(m_reductionArray, 0, m_width * m_height * sizeof(float)) );
	
	// Compute prior function value and gradient without final filtering
	huber<<<gridDim, blockDim>>>(d_tmp, m_width, m_height, m_alpha, m_strength, pixelsPerThread, m_reductionArray);

	CudaCheckError();
	hipDeviceSynchronize();
	
	Reduction::sumReduction(m_reductionArray, m_width, m_height, m_width, d_f, m_reductionArray2);
	
#ifdef SUPERRES_TIMING
	m_atomic->stop();
	m_filter->start();
#endif

	// Compute Laplacian of the gradient
	laplacian<<<gridDim, blockDim>>>(d_gradf, d_tmp, m_width, m_height, pixelsPerThread);

	CudaCheckError();
	hipDeviceSynchronize();
	
#ifdef SUPERRES_TIMING
	m_filter->stop();
#endif
}


namespace gpu_HuberLaplacian
{

	__global__ void laplacian(float *dst, const float *src, const size_t width, const size_t height, 
	                    const size_t pixelsPerThread)
	{
		const size_t col  = (blockIdx.x * blockDim.x + threadIdx.x) % width;
		const size_t crow = (blockIdx.x * blockDim.x + threadIdx.x) / width * pixelsPerThread;
		
		if (col >= width || crow >= height)
			return;

		const size_t srow = crow + 1;
		const size_t erow = min((unsigned int)(crow + pixelsPerThread - 1), (unsigned int)(height - 1));
		
		// First element

		const size_t firstIdx = crow * width + col;

		dst[firstIdx] = src[firstIdx];
		
		if (crow + 1 <  height) dst[firstIdx] -= 0.25f * src[firstIdx + width]; // S
		if (crow     >= 1)      dst[firstIdx] -= 0.25f * src[firstIdx - width]; // N
		if (col + 1  <  width)  dst[firstIdx] -= 0.25f * src[firstIdx + 1]; // E
		if (col      >= 1)      dst[firstIdx] -= 0.25f * src[firstIdx - 1]; // W

		// Inner elements

		for (int row = srow; row < erow; ++row)
		{
			const size_t cIdx = row * width + col;
		
			// C, S, N (always exist)
			dst[cIdx] = src[cIdx] - 0.25f * (src[cIdx + width] + src[cIdx - width]);

			if (col + 1 < width) dst[cIdx] -= 0.25f * src[cIdx + 1]; // E
			if (col     >= 1)    dst[cIdx] -= 0.25f * src[cIdx - 1]; // W
		}
		
		if (erow <= crow)
			return;

		// Last element

		const size_t lastIdx = erow * width + col;

		dst[lastIdx] = src[lastIdx] - 0.25f * src[lastIdx - width]; // C, N
		
		if (erow + 1 <  height) dst[lastIdx] -= 0.25f * src[lastIdx + width]; // S
		if (col + 1  <  width)  dst[lastIdx] -= 0.25f * src[lastIdx + 1]; // E
		if (col      >= 1)      dst[lastIdx] -= 0.25f * src[lastIdx - 1]; // W
	}

	__global__ void huber(float *a, const size_t width, const size_t height, const float alpha, 
	                      const float strength, const size_t pixelsPerThread, float *f)
	{
		const size_t col  = (blockIdx.x * blockDim.x + threadIdx.x) % width;
		const size_t crow = (blockIdx.x * blockDim.x + threadIdx.x) / width * pixelsPerThread;
		
		if (col >= width || crow >= height)
			return;

		const size_t erow = min((unsigned int)(crow + pixelsPerThread), (unsigned int)height);

		const float alpha2 = alpha * alpha;

		float colF = 0.0f;

		for (size_t row = crow; row < erow; ++row)
		{
			const size_t idx = row * width + col;
		
			// Pseudo-Huber loss function
			const float root = sqrtf(1.0f + a[idx]*a[idx] / alpha2); 
			colF += alpha2 * (root - 1.0f);
			a[idx] *= strength / root;
		}

		colF *= strength;
		f[blockIdx.x * blockDim.x + threadIdx.x] = colF;
	}

}
